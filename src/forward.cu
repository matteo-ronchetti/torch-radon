#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "floatcast.h"
#include "utils.h"
#include "texture.h"
#include "parameter_classes.h"
#include "log.h"


template<bool parallel_beam, int channels, typename T>
__global__ void
radon_forward_kernel(T *__restrict__ output, hipTextureObject_t texture, const float *__restrict__ angles,
                     const VolumeCfg vol_cfg, const ProjectionCfg proj_cfg) {

    // Calculate texture coordinates
    const int ray_id = blockIdx.x * blockDim.x + threadIdx.x;
    const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;

    const int base = ray_id + proj_cfg.det_count_u * (angle_id + proj_cfg.n_angles * blockIdx.z);
    const int mem_pitch = proj_cfg.det_count_u * proj_cfg.n_angles * blockDim.z * gridDim.z;

    if (angle_id < proj_cfg.n_angles && ray_id < proj_cfg.det_count_u) {
        float accumulator[channels];

#pragma unroll
        for (int i = 0; i < channels; i++) accumulator[i] = 0.0f;

        // compute ray
        float sx, sy, ex, ey;
        if (parallel_beam) {
            sx = (ray_id - proj_cfg.det_count_u * 0.5f + 0.5f) * proj_cfg.det_spacing_u;
            sy = hypot(abs(vol_cfg.dx) + vol_cfg.width * 0.5f, abs(vol_cfg.dy) + vol_cfg.height * 0.5f);
            ex = sx;
            ey = -sy;
        } else {
            sy = proj_cfg.s_dist;
            sx = 0.0f;
            ey = -proj_cfg.d_dist;
            ex = (ray_id - proj_cfg.det_count_u * 0.5f + 0.5f) * proj_cfg.det_spacing_u;
        }

        // rotate ray
        const float angle = angles[angle_id];
        const float cs = __cosf(angle);
        const float sn = __sinf(angle);

        // start position rs and direction rd (in detector coordinate system)
        float rsx = sx * cs + sy * sn;
        float rsy = -sx * sn + sy * cs;
        float rdx = ex * cs + ey * sn - rsx;
        float rdy = -ex * sn + ey * cs - rsy;

        // convert coordinates to volume coordinate system
        const float vol_orig_x = vol_cfg.dx - 0.5f * vol_cfg.width * vol_cfg.sx;
        const float vol_orig_y = vol_cfg.dy - 0.5f * vol_cfg.height * vol_cfg.sy;
        rsx = (rsx - vol_orig_x) * vol_cfg.inv_scale_x; 
        rsy = (rsy - vol_orig_y) * vol_cfg.inv_scale_y; 
        rdx = rdx * vol_cfg.inv_scale_x; 
        rdy = rdy * vol_cfg.inv_scale_y; 


        // clip to volume (to reduce memory reads)
        float dx = rdx >= 0 ? max(rdx, 1e-6) : min(rdx, -1e-6);
        float dy = rdy >= 0 ? max(rdy, 1e-6) : min(rdy, -1e-6);

        const float alpha_x_m = (- rsx) / dx;
        const float alpha_x_p = (vol_cfg.width - rsx) / dx;
        const float alpha_y_m = (- rsy) / dy;
        const float alpha_y_p = (vol_cfg.height - rsy) / dy;
        const float alpha_s = max(min(alpha_x_p, alpha_x_m), min(alpha_y_p, alpha_y_m));
        const float alpha_e = min(max(alpha_x_p, alpha_x_m), max(alpha_y_p, alpha_y_m));

        // if ray volume intersection is empty exit
        if (alpha_s > alpha_e) {
#pragma unroll
            for (int b = 0; b < channels; b++) output[base + b * mem_pitch] = toType<T>(0.0f);
            return;
        }

        rsx += rdx * alpha_s;
        rsy += rdy * alpha_s;
        rdx *= (alpha_e - alpha_s);
        rdy *= (alpha_e - alpha_s);

        const int n_steps = __float2int_rn(max(abs(rdx), abs(rdy)));
        const float vx = rdx / max(abs(rdx), abs(rdy));
        const float vy = rdy / max(abs(rdx), abs(rdy));
        const float n = hypot(vx * vol_cfg.sx, vy * vol_cfg.sy);
        
        float step;
        if(abs(rdy) >= abs(rdx)){
            float y_increment = 0.5f - rsy + __float2int_rn(rsy);
            step = y_increment / vy;
            step += vy < 0;
        }else{
            float x_increment = 0.5f - rsx + __float2int_rn(rsx);
            step = x_increment / vx;
            step += vx < 0;
        }
        rsx += step*vx;
        rsy += step*vy;

        #pragma unroll(4)
        for (int j = 0; j < n_steps; j++) {
            if (channels == 1) {
                accumulator[0] += tex2DLayered<float>(texture, rsx, rsy, blockIdx.z);
            } else {
                float4 read = tex2DLayered<float4>(texture, rsx, rsy, blockIdx.z);

                accumulator[0] += read.x;
                accumulator[1] += read.y;
                accumulator[2] += read.z;
                accumulator[3] += read.w;
            }
            rsx += vx;
            rsy += vy;
        }
        
        #pragma unroll
        for (int b = 0; b < channels; b++) output[base + b * mem_pitch] = toType<T>(accumulator[b] * n);
    }
}

template<typename T>
void radon_forward_cuda(
        const T *x, const float *angles, T *y, TextureCache &tex_cache,
        const VolumeCfg &vol_cfg, const ProjectionCfg &proj_cfg, const ExecCfg &exec_cfg,
        const int batch_size, const int device
) {
    constexpr bool is_float = std::is_same<T, float>::value;
    constexpr int precision = is_float ? PRECISION_FLOAT : PRECISION_HALF;
    const int channels = exec_cfg.get_channels(batch_size);

    // copy x into CUDA Array (allocating it if needed) and bind to texture
    Texture *tex = tex_cache.get(
            {device, batch_size / channels, vol_cfg.height, vol_cfg.width, true, channels, precision});
    tex->put(x);

    // Invoke kernel
    const dim3 grid_dim = exec_cfg.get_grid_size(proj_cfg.det_count_u, proj_cfg.n_angles, batch_size / channels);
    const dim3 block_dim = exec_cfg.get_block_dim();

    LOG_DEBUG("Block Size x:" << block_dim.x << " y:" << block_dim.y << " z:" << block_dim.z);
    LOG_DEBUG("Grid Size x:" << grid_dim.x << " y:" << grid_dim.y << " z:" << grid_dim.z);

    if (proj_cfg.projection_type == FANBEAM) {
        if (channels == 1) {
            radon_forward_kernel<false, 1> << < grid_dim, block_dim >> >
                                                                     ((float*)y, tex->texture, angles, vol_cfg, proj_cfg);
        } else {
            if (is_float) {
                radon_forward_kernel<false, 4> << < grid_dim, block_dim >> >
                                                                         ((float*)y, tex->texture, angles, vol_cfg, proj_cfg);
            } else {
                radon_forward_kernel<false, 4> << < grid_dim, block_dim >> >
                                                                         ((__half *) y, tex->texture, angles, vol_cfg, proj_cfg);
            }
        }
    } else {
        if (channels == 1) {
            radon_forward_kernel<true, 1> << < grid_dim, block_dim >> >
                                                                    ((float*)y, tex->texture, angles, vol_cfg, proj_cfg);
        } else {
            if (is_float) {
                radon_forward_kernel<true, 4> << < grid_dim, block_dim >> >
                                                                        ((float*)y, tex->texture, angles, vol_cfg, proj_cfg);
            } else {
                radon_forward_kernel<true, 4> << < grid_dim, block_dim >> >
                                                                        ((__half *) y, tex->texture, angles, vol_cfg, proj_cfg);
            }
        }
    }
}

template void
radon_forward_cuda<float>(const float *x, const float *angles, float *y, TextureCache &tex_cache,
                          const VolumeCfg &vol_cfg, const ProjectionCfg &proj_cfg, const ExecCfg &exec_cfg,
                          const int batch_size, const int device);

template void radon_forward_cuda<unsigned short>(const unsigned short *x, const float *angles, unsigned short *y,
                                                 TextureCache &tex_cache,
                                                 const VolumeCfg &vol_cfg, const ProjectionCfg &proj_cfg,
                                                 const ExecCfg &exec_cfg,
                                                 const int batch_size, const int device);


template<int channels, typename T>
__global__ void
radon_forward_kernel_3d(T *__restrict__ output, hipTextureObject_t texture, const float *__restrict__ angles,
                        const VolumeCfg vol_cfg, const ProjectionCfg proj_cfg) {
    // Calculate sensor coordinates in pixels
    // TODO is there an "optimal" map from thread to coordinates that maximizes cache hits?
    // TODO check other permutations (combined with different block sizes)
    const int pu = blockIdx.x * blockDim.x + threadIdx.x;
    const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;
    const int pv = blockIdx.z * blockDim.z + threadIdx.z;

    const uint index = (angle_id * proj_cfg.det_count_v + pv) * proj_cfg.det_count_u + pu;
    const uint mem_pitch = proj_cfg.n_angles * proj_cfg.det_count_v * proj_cfg.det_count_u;

    if (angle_id < proj_cfg.n_angles && pu < proj_cfg.det_count_u && pv < proj_cfg.det_count_v) {
        // define accumulator
        float accumulator[channels];
#pragma unroll
        for (int i = 0; i < channels; i++) accumulator[i] = 0.0f;

        // compute ray
        const float angle = angles[angle_id];
        const float cs = __cosf(angle);
        const float sn = __sinf(angle);

        float sx = 0.0f;
        float sy = -proj_cfg.s_dist;
        // sz = initial_z + pitch * angle / (2*pi);
        float rsz = proj_cfg.initial_z + proj_cfg.pitch * angle * 0.1591549f;

        float ex = (pu - proj_cfg.det_count_u * 0.5f + 0.5f) * proj_cfg.det_spacing_u;
        float ey = proj_cfg.d_dist;
        // z is not affected by rotation
        float rdz = (pv - proj_cfg.det_count_v * 0.5f + 0.5f) * proj_cfg.det_spacing_v;

        // rotate start position rs and direction rd
        float rsx = sx * cs - sy * sn;
        float rsy = sx * sn + sy * cs;
        float rdx = ex * cs - ey * sn - rsx;
        float rdy = ex * sn + ey * cs - rsy;

        // convert coordinates to volume coordinate system
        const float vol_orig_x = vol_cfg.dx - 0.5f * vol_cfg.width * vol_cfg.sx;
        const float vol_orig_y = vol_cfg.dy - 0.5f * vol_cfg.height * vol_cfg.sy;
        const float vol_orig_z = vol_cfg.dz - 0.5f * vol_cfg.depth * vol_cfg.sz;
        rsx = (rsx - vol_orig_x) * vol_cfg.inv_scale_x; 
        rsy = (rsy - vol_orig_y) * vol_cfg.inv_scale_y; 
        rsz = (rsz - vol_orig_z) * vol_cfg.inv_scale_z; 
        rdx = rdx * vol_cfg.inv_scale_x; 
        rdy = rdy * vol_cfg.inv_scale_y;
        rdz = rdz * vol_cfg.inv_scale_z;

        // Clip ray to cube to reduce the number of memory reads
        float dx = rdx >= 0 ? max(rdx, 1e-6) : min(rdx, -1e-6);
        float dy = rdy >= 0 ? max(rdy, 1e-6) : min(rdy, -1e-6);
        float dz = rdz >= 0 ? max(rdz, 1e-6) : min(rdz, -1e-6);

        const float alpha_x_m = (- rsx) / dx;
        const float alpha_x_p = (vol_cfg.width - rsx) / dx;
        const float alpha_y_m = (- rsy) / dy;
        const float alpha_y_p = (vol_cfg.height - rsy) / dy;
        const float alpha_z_m = (- rsz) / dz;
        const float alpha_z_p = (vol_cfg.depth - rsz) / dz;

        const float alpha_s = max(min(alpha_x_p, alpha_x_m), max(min(alpha_y_p, alpha_y_m), min(alpha_z_p, alpha_z_m)));
        const float alpha_e = min(max(alpha_x_p, alpha_x_m), min(max(alpha_y_p, alpha_y_m), max(alpha_z_p, alpha_z_m)));

        if (alpha_s > alpha_e) {
#pragma unroll
            for (int b = 0; b < channels; b++) output[b * mem_pitch + index] = toType<T>(0.0f);
            return;
        }

        rsx += rdx * alpha_s;
        rsy += rdy * alpha_s;
        rsz += rdz * alpha_s;
        rdx *= (alpha_e - alpha_s);
        rdy *= (alpha_e - alpha_s);
        rdz *= (alpha_e - alpha_s);

        // accumulate loop
        const float f_n_steps = max(abs(rdx), max(abs(rdy), abs(rdz)));
        const int n_steps = __float2uint_ru(f_n_steps);
        const float vx = rdx / f_n_steps;
        const float vy = rdy / f_n_steps;
        const float vz = rdz / f_n_steps;
        const float n = norm3df(vx * vol_cfg.sx, vy * vol_cfg.sy, vz * vol_cfg.sz);
        
        float step;
        if(abs(rdy) >= abs(rdx)){
            float y_increment = 0.5f - rsy + __float2int_rn(rsy);
            step = y_increment / vy;
            step += vy < 0;
        }else{
            float x_increment = 0.5f - rsx + __float2int_rn(rsx);
            step = x_increment / vx;
            step += vx < 0;
        }
        rsx += step*vx;
        rsy += step*vy;
        rsz += step*vz;

        #pragma unroll(4)
        for (int j = 0; j <= n_steps; j++) {
            if (channels == 1) {
                accumulator[0] += tex3D<float>(texture, rsx, rsy, rsz);
            } else {
                float4 read = tex3D<float4>(texture, rsx, rsy, rsz);
                accumulator[0] += read.x;
                accumulator[1] += read.y;
                accumulator[2] += read.z;
                accumulator[3] += read.w;
            }

            rsx += vx;
            rsy += vy;
            rsz += vz;
        }

        // output
#pragma unroll
        for (int b = 0; b < channels; b++) {
            output[b * mem_pitch + index] = toType<T>(accumulator[b] * n);
        }
    }
}

template<typename T>
void radon_forward_cuda_3d(
        const T *x, const float *angles, T *y, TextureCache &tex_cache,
        const VolumeCfg &vol_cfg, const ProjectionCfg &proj_cfg, const ExecCfg &exec_cfg,
        const int batch_size, const int device
) {
    constexpr bool is_float = std::is_same<T, float>::value;
    constexpr int precision = is_float ? PRECISION_FLOAT : PRECISION_HALF;
    const int channels = exec_cfg.get_channels(batch_size);

    Texture *tex = tex_cache.get(
            {device, vol_cfg.depth, vol_cfg.height, vol_cfg.width, false, channels, precision});

    const dim3 grid_dim = exec_cfg.get_grid_size(proj_cfg.det_count_u, proj_cfg.n_angles, proj_cfg.det_count_v);
    const dim3 block_dim = exec_cfg.get_block_dim();


    for (int i = 0; i < batch_size; i += channels) {
        T *local_y = &y[i * proj_cfg.det_count_u * proj_cfg.det_count_v * proj_cfg.n_angles];
        tex->put(&x[i * vol_cfg.depth * vol_cfg.height * vol_cfg.width]);

        // Invoke kernel
        if (channels == 1) {
            radon_forward_kernel_3d<1> << < grid_dim, block_dim >> >
                                                      (local_y, tex->texture, angles, vol_cfg, proj_cfg);
        } else {
            if (is_float) {
                radon_forward_kernel_3d<4> << < grid_dim, block_dim >> >
                                                          (local_y, tex->texture, angles, vol_cfg, proj_cfg);
            } else {
                radon_forward_kernel_3d<4> << < grid_dim, block_dim >> >
                                                          ((__half *) local_y, tex->texture, angles, vol_cfg, proj_cfg);
            }
        }
    }
}

template void
radon_forward_cuda_3d<float>(const float *x, const float *angles, float *y, TextureCache &tex_cache,
                             const VolumeCfg &vol_cfg, const ProjectionCfg &proj_cfg, const ExecCfg &exec_cfg,
                             const int batch_size, const int device);

template void radon_forward_cuda_3d<unsigned short>(const unsigned short *x, const float *angles, unsigned short *y,
                                                    TextureCache &tex_cache,
                                                    const VolumeCfg &vol_cfg, const ProjectionCfg &proj_cfg,
                                                    const ExecCfg &exec_cfg,
                                                    const int batch_size, const int device);